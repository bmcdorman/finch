
#include <hip/hip_runtime.h>
#ifdef USE_CUDA

#include <finch/cuda/cuda_evaluator.hpp>

#include "program_interpreter.hpp"

# warning remove
#include <iostream>

using namespace finch;

cuda_evaluator::cuda_evaluator()
{
  
}

void cuda_evaluator::evaluate(const matrix2<uint16_t> &maze, population &generation,
  const program_state &initial_state, const uint32_t op_lim) const
{
  using namespace std;
  vector<uint32_t> offsets;
  vector<uint32_t> programs;
  population::iterator git = generation.begin();
  for(; git != generation.end(); ++git) {
    const agent &current = *git;
    offsets.push_back(programs.size());
    current.program().write(programs);
  }

  const size_t maze_size = sizeof(uint16_t) * maze.rows() * maze.columns();
  uint16_t *device_maze = 0;
  cudaMalloc(&device_maze, maze_size);
  const uint16_t *const maze_data = maze.ptr();
  cudaMemcpy(device_maze, maze_data, maze_size, cudaMemcpyHostToDevice);
  
  vector<program_state> res(generation.size());
  
  const size_t programs_size = programs.size() * sizeof(uint32_t);
  uint32_t *device_programs = 0;
  cudaMalloc(&device_programs, programs_size);
  cudaMemcpy(device_programs, &programs[0], programs_size, cudaMemcpyHostToDevice);
  
  const size_t offsets_size = offsets.size() * sizeof(uint32_t);
  uint32_t *device_offsets = 0;
  cudaMalloc(&device_offsets, offsets_size);
  cudaMemcpy(device_offsets, &offsets[0], offsets_size, cudaMemcpyHostToDevice);
  
  
  const size_t res_size = generation.size() * sizeof(program_state);
  program_state *device_res = 0;
  cudaMalloc(&device_res, res_size);
  
  const size_t total_size = programs_size + offsets_size + res_size;
  
  cout << "Allocating " << static_cast<double>(total_size) / 1024.0 << " KiB on GPU" << endl;
  
  // FIXME: Rounding error
  uint32_t threads = 10;
  uint32_t blocks = ceil((float)generation.size() / threads);
  cudaEvent_t start_event;
  cudaEvent_t end_event;
  cudaEventCreate(&start_event);
  cudaEventCreate(&end_event);
  cudaEventRecord(start_event, 0);
  program_interpreter<<<blocks, threads>>>(device_maze, maze.rows(), maze.columns(),
    device_offsets, device_programs, initial_state, op_lim, device_res);
  cudaEventRecord(end_event, 0);
  
  cudaEventSynchronize(start_event);
  cudaEventSynchronize(end_event);
  
  float dt_ms;
  cudaEventElapsedTime(&dt_ms, start_event, end_event);
  
  cudaEventDestroy(start_event);
  cudaEventDestroy(end_event);
  
  // cout << "Time: " << dt_ms << " ms" << endl;
  
  cudaError_t err = cudaSuccess;
  if((err = cudaGetLastError()) != cudaSuccess) {
    cerr << "CUDA Error: " << cudaGetErrorString(err) << endl;
  }
  
  cudaMemcpy(&res[0], device_res, res_size, cudaMemcpyDeviceToHost);
  
  git = generation.begin();
  for(vector<program_state>::const_iterator it = res.begin();
    it != res.end() && git != generation.end(); ++it, ++git)
  {
    (*git).set_final_state(*it);
  }
  
  cudaFree(device_res);
  cudaFree(device_offsets);
  cudaFree(device_programs);
  cudaFree(device_maze);
}

#endif