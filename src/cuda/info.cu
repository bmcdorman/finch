#include <cuda/info.hpp>
#include <iostream>

using namespace std;

void print_cuda_info()
{
  int gpu = 0;
  hipDeviceProp_t gpuProp;
  hipGetDevice(&gpu);
  hipGetDeviceProperties(&gpuProp, gpu);
  cout << "GPU Information:" << endl;
  cout << "\tName: " << gpuProp.name << "(" << gpuProp.major << "." << gpuProp.minor << ")" << endl;
  cout << "\tMemory: " << gpuProp.totalGlobalMem << endl;
  cout << "\tMax Threads/Block: " << gpuProp.maxThreadsPerBlock << endl;
  cout << "\tECC Enabled: " << gpuProp.eccEnabled << endl;
}